/*USAGE: compara_kernels –N=<dim_mat> -W=<dim_bloq> -K=<kernel>*/
/*KERNEL VALUES:*/
#define SIMPLY_MULTIPLY 1
#define COALESCED_MULTIPLY 2
#define SHARED_AB_MULTIPLY 3

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

// includes, project
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>

// These are CUDA Helper functions for initialization and error checking
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

#include "utils.h"

////////////////////////////////////////////////////////////////////////////////
// kernel includes
/*kernels: simpleMultiply, sharedABMultiply, coalescedMultiply
common header for all kernels: (float *a, float* b, float *c,int N)*/
#include "./kernels/simpleMultiply_kernel.cu"
#include "./kernels/sharedABMultiply_kernel.cu"
#include "./kernels/coalescedMultiply_kernel.cu"
////////////////////////////////////////////////////////////////////////////////

/**************REQUIREMENTS**********************/
/*Threads per Block: WxW (the number of threads per block will be the leading dimension of the tile)*/
/*Tile size: WxW*/
/*N is a multiple of W so N=WxT with T as int*/
/*Grid size: TxT*/
/*Each thread calculates one and only one element of C using a row and column of b (kernels already built like that)*/
/*Each thread block calculates the elements from a tile by multiplying a tile from A with a tile from B*/


//kernel_runner function to run kernel to specific user specs and record time in start and stop
float kernel_run_chrono(int N, int kernel, dim3 grid, dim3 block, float *A_d, float *B_d, float *C_d, int tile_dim)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    switch (kernel){
        case SIMPLY_MULTIPLY:
            //shared memory size set to simpleMultiply tile: 0
            simpleMultiply<<<grid, block>>>(A_d, B_d, C_d, N);
            break;
        case COALESCED_MULTIPLY:
            //shared memory size set to coalescedMultiply tile: a_tile_dims * sizeof(float) -> tile_dim * tile_dim * sizeof(float)
            coalescedMultiply<<<grid, block, tile_dim * tile_dim * sizeof(float) >>>(A_d, B_d, C_d, N);
            break;
        case SHARED_AB_MULTIPLY:
            //shared memory size set to sharedABMultiply tile: 2 * a_tile_dims * sizeof(float) -> 2 * tile_dim * tile_dim * sizeof(float)
            sharedABMultiply<<<grid, block, 2 * tile_dim * tile_dim * sizeof(float)>>>(A_d, B_d, C_d, N);
            break;
        default:
            printf("Invalid kernel\n");
            return 1;
    }

    // 6.2. Stop GPU timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGPU;
    hipEventElapsedTime(&elapsedTimeGPU, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTimeGPU;

}

//run_program function to run program to specific user specs
int run_test(int N, int W, int kernel)
{
    int T = N/W;
    dim3 grid(T, T); //Grid size: TxT
    dim3 block(W, W); //Threads per Block: WxW
    int tile_dim = W; //Tile size: WxW

    /*2. Allocate memory and initialiaze for matrices A, B and C*/
    //initialize_matrices(int M, int K, int N, float** A, float** B, float** C) 
    float *A, *B, *C;
    initialize_matrices(N, W, N, &A, &B, &C);

    /*4. Allocate memory for matrices A_d, B_d and C_d in the device*/
    float *A_d, *B_d, *C_d;
    //allocate_device_matrices(int M_gpu, int K, int N, float** A_d, float** B_d, float** C_d)
    allocate_device_matrices(N, W, N, &A_d, &B_d, &C_d);

    /*5. Copy matrices A and B to the device*/
    // matrix_device_copy(float *A, float *B, float *A_d, float *B_d, int N, int W)
    matrix_device_copy(A, B, A_d, B_d, N, W);

    /*6. Call the kernel that the user has chosen*/
    
    // 6.1. Start GPU timer
    //kernel run chrono: run kernel to specific user specs leaving the result in C_d and record time in start and stop
    kernel_run_chrono(N, kernel, grid, block, A_d, B_d, C_d, tile_dim);

    /*7. Copy matrix C from the device to the host*/
    checkCudaErrors(hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost));

    /*8. Check the result using a CPU program*/
    //int matrix_check(float *A, float *B, float *C, int N, int W, int kernel)
    int result = matrix_check(A, B, C, N, W, kernel);
    
    //if the test passed, display the configuration (N, W, kernel)
    if(result == 0)
    {
        printf("TEST PASSED: N=%d, W=%d, K=%d\n", N, W, kernel);
    }
    else
    {
        printf("TEST FAILED: N=%d, W=%d, K=%d\n", N, W, kernel);
        exit(EXIT_FAILURE);
    }

    /*9. Free memory*/
    free(A);
    free(B);
    free(C);
    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d));

    return 0;
}

void exhaustive_test()
{
    //We must comply with the restrictions imposed by the problem in the tests:
    //Threads per Block: WxW (the number of threads per block will be the leading dimension of the tile)
    //Tile size: WxW
    //N is a multiple of W so N=WxT with T as int
    //Grid size: TxT

    //Values:
    // N = [1, 4096]
    // W = [1, 32]
    // K = [1, 3]

    //n_max up to 4096 and w_max up to 32
    int n_max = 256;
    int w_max = 32;
    int kernel = 1;
    int kernel_passed[3] = {0, 0, 0}; //Array to store if a kernel passed all tests


    //We will test all possible (allowed by the restrictions) combinations of N, W and K
    int passed_tests = 0;
    int failed_tests = 0;

    //Loop for kernels 
    for(kernel = 1; kernel < 4; kernel ++)
    {
        passed_tests = 0;
        failed_tests = 0;
        //Loop for N
        for(int n = 1; n <= n_max; n++)
        {
            //Loop for W
            for(int w = 1; w <= w_max; w++)
            {
                //Check if N is a multiple of W
                if(n % w == 0)
                {
                    //Run test
                    if(run_test(n, w, kernel) == 0)
                    {
                        passed_tests++;
                    }
                    else
                    {
                        failed_tests++;
                    }
                }
            }
        }
        //display results for each kernel
        printf("##################KERNEL %d RESULTS###############\n", kernel);
        printf("PASSED TESTS: %d\n", passed_tests);
        printf("FAILED TESTS: %d\n", failed_tests);
        printf("###############################################\n");
        if(failed_tests == 0)
        {
            kernel_passed[kernel - 1] = 1;
        }
    }
    //display final results
    printf("##################FINAL RESULTS###############\n");
    for(int i = 0; i < 3; i++)
    {
        if(kernel_passed[i] == 1)
        {
            printf("KERNEL %d PASSED ALL TESTS\n", i + 1);
        }
        else
        {
            printf("KERNEL %d FAILED SOME TESTS\n", i + 1);
        }
    }
    printf("###############################################\n");
    
}

void exhaustive_bench()
{
    /*N={512,1024,2048,4096} y W={4,8,16,32}*/
    int N[4] = {512, 1024, 2048, 4096};
    int W[4] = {4, 8, 16, 32};
    int kernel[3] = {SIMPLY_MULTIPLY, COALESCED_MULTIPLY, SHARED_AB_MULTIPLY};

    //loop through all possible combinations (first kernel, then N, then W)
    for (int i = 0; i < 3; i++){
        for (int j = 0; j < 4; j++){
            for (int k = 0; k < 4; k++){
                run_test(N[j], W[k], kernel[i]);
            }
        }
    }
    
}

//main function
int main(int argc, char **argv){
    exhaustive_test();
    //exhaustive_bench();
}