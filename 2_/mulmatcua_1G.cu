/*USAGE: compara_kernels -N=<dim_mat> -W=<dim_bloq> -K=<kernel>*/
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#define DEBUG

// includes, project
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>

// These are CUDA Helper functions for initialization and error checking
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

#include "utils.h"

////////////////////////////////////////////////////////////////////////////////
// kernel includes
/*kernels: simpleMultiply, sharedABMultiply, coalescedMultiply
common header for all kernels: (float *a, float* b, float *c,int N)*/
#include "sharedABMultiply_kernel_super_tile.cu"
////////////////////////////////////////////////////////////////////////////////

/**************REQUIREMENTS**********************/
/*Threads per Block: WxW (the number of threads per block will be the leading dimension of the tile)*/
/*Tile size: WxW*/
/*N is a multiple of W so N=WxT with T as int*/
/*Grid size: TxT*/
/*Each thread calculates one and only one element of C using a row and column of b (kernels already built like that)*/
/*Each thread block calculates the elements from a tile by multiplying a tile from A with a tile from B*/

//kernel_runner function to run kernel to specific user specs and record time in start and stop
float kernel_run_chrono(int N, dim3 grid, dim3 block, float *A_d, float *B_d, float *C_d, int tile_dim)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    sharedABMultiply<<<grid, block, 2 * tile_dim * tile_dim * sizeof(float)>>>(A_d, B_d, C_d, N, tile_dim);

    // 6.2. Stop GPU timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGPU;
    hipEventElapsedTime(&elapsedTimeGPU, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTimeGPU;

}

//run_program function to run program to specific user specs
int run_test(int N, int W)
{
    int T = N/W;
    dim3 grid(T, T); //Grid size: TxT
    dim3 block(W, W); //Threads per Block: WxW
    int tile_dim = W; //Tile size: WxW

    /*2. Allocate memory and initialiaze for matrices A, B and C*/
    //initialize_matrices(int M, int K, int N, float** A, float** B, float** C) 
    float *A, *B, *C;
    initialize_matrices(N, N, N, &A, &B, &C);

    /*4. Allocate memory for matrices A_d, B_d and C_d in the device*/
    float *A_d, *B_d, *C_d;
    //allocate_device_matrices(int M_gpu, int K, int N, float** A_d, float** B_d, float** C_d)
    allocate_device_matrices(N, N, N, &A_d, &B_d, &C_d);

    /*5. Copy matrices A and B to the device*/
    // matrix_device_copy_mnk(float *A, float *B, float *A_d, float *B_d, int M, int N, int K)
    matrix_device_copy_mnk(A, B, A_d, B_d, N, N, N);

    /*6. Call the kernel that the user has chosen*/
    float elapsedTimeGPU = kernel_run_chrono(N, grid, block, A_d, B_d, C_d, tile_dim);

    /*7. Copy matrix C from the device to the host*/
    checkCudaErrors(hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost));

    /*8. Print the execution time of the kernel*/
    //8.1 print Header
    printf("N, W, Time\n");
    //8.2 print values
    printf("%d, %d, %f\n", N, W, elapsedTimeGPU);

    /*EXTRA: Print all Matrices */
    printf("Matrix A:\n");
    print_matrix(A, N, N);
    printf("\n");
    printf("Matrix B:\n");
    print_matrix(B, N, N);
    printf("\n");
    printf("Matrix C:\n");
    print_matrix(C, N, N);
    printf("\n");

    /*9. Check the result using a CPU program*/
    //int matrix_check_v2(float *A, float *B, float *C, int N, int W)
    #ifdef DEBUG
    int result = matrix_check_v2(A, B, C, N, W);
    if(result == 0)
    {
        printf("RESULT OK: N=%d, W=%d\n", N, W);
    }
    else
    {
        printf("RESULT FAILED: N=%d, W=%d\n", N, W);
    }
    #endif


    /*9. Free memory*/
    free(A);
    free(B);
    free(C);
    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d));

    return 0;
}


//main function
int main(int argc, char **argv){
    /*1. parse arguments: compara_kernels -N=<dim_mat> -W=<dim_bloq>*/ 
    int N, W;
    if (argc != 3){
        printf("Usage: compara_kernels -N=<dim_mat> -W=<dim_bloq>\n");
        return 1;
    }
    sscanf(argv[1], "-N=%d", &N);
    sscanf(argv[2], "-W=%d", &W);
    if (N % W != 0){
        printf("N must be a multiple of W\n");
        return 1;
    }
    if (W > 32){
        printf("W must be less than 32\n");
        return 1;
    }
    run_test(N, W);
    
}